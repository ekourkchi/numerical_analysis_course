
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void VecEval(float *C)
{
  int i = threadIdx.x;
  int j;
  for (j=0; j<=100; j++)
    C[i] = 1./sqrt(1.+i) + j*sin(2.*sin(0.1*i));
}

int main()
{ const int N=1000000;
  float C[N];
  // Kernel invocation with N threads 
  VecEval<<<1, N>>>(C);
}
